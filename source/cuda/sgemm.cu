#include "hip/hip_runtime.h"
#include "sgemm.cuh"
#include <chrono>

namespace fastnum {
namespace cuda {
namespace kernel {

// #define SMEM_LDA (128)
// #define SMEM_LDB (128)

// __global__ void sgemm_nn(int M, int N, int K, 
//                          float alpha,
//                          const float* A, int lda,
//                          const float* B, int ldb,
//                          float beta,
//                          float* C, int ldc) {

//     __shared__ float a_shared[128 * 8];
//     __shared__ float b_shared[128 * 8];

//     const float* a_ptr = A + blockIdx.y * 128 * lda;
//     const float* b_ptr = B + blockIdx.x * 128;


//     int sac = (threadIdx.x / 8) * 4; 
//     int sar = (threadIdx.x % 8);
//     int sbr = threadIdx.x / 32;
//     int sbc = threadIdx.x % 32;


//     float a_penal[8] = {0.0f};
//     float b_penal[8] = {0.0f};
//     float c_penal[8][8] = {0.0f};

//     for(int k=0; k < K; k+=8) {
// #pragma unroll
//         for(int x=0; x < 4; ++x) {
//             if (blockIdx.y * 128 + sac + x < M && sar + k < K) {
//                 a_shared[sar * 128 + sac + x] = alpha * a_ptr[(sac + x) * lda + sar];
//             } else {
//                 a_shared[sar * 128 + sac + x] = 0.0f;
//             }
//         } 
// #pragma unroll
//         for(int x=0; x < 4; ++x) {
//             if (sbr + k < K && sbc + x * 32 + blockIdx.x * 128 < N) {
//                 b_shared[sbr * 128 + sbc + x * 32]  = b_ptr[sbr * ldb + sbc + x * 32];
//             } else {
//                 b_shared[sbr * 128 + sbc + x * 32]  = 0.0f;
//             }
//         }

//         __syncthreads();

//         a_ptr += 8;
//         b_ptr += 8*ldb;

//         int apc = (threadIdx.x / 16) * 4;
//         int bpc = (threadIdx.x % 16) * 4;
// #pragma unroll
//         for(int subk = 0; subk < 8; ++subk) {
//             int skc = subk * 128;
// #pragma unroll
//             for(int x=0; x < 4; ++x) {
//                 a_penal[x] = a_shared[skc + apc + x]; 
//                 a_penal[x + 4] = a_shared[skc + apc + x + 64]; 
//             }
// #pragma unroll
//             for(int x=0; x < 4; ++x) {
//                 b_penal[x] = b_shared[skc + bpc + x];
//                 b_penal[x + 4] = b_shared[skc + bpc + x + 64];
//             }

// #pragma unroll
//             for(int y=0; y < 8; ++y) {
// #pragma unroll
//                 for(int x=0; x < 8; ++x) {
//                     c_penal[y][x] += a_penal[y] * b_penal[x];
//                 }
//             } 
//         }
//         __syncthreads();
//     }

//     int offset_y = blockIdx.y * 128 +  (threadIdx.x / 16) * 4;
//     int offset_x = blockIdx.x * 128 + (threadIdx.x % 16) * 4;
//     int c_offset = offset_y * ldc + offset_x;
//     float* c_ptr = C + c_offset;

// #pragma unroll
//     for(int y=0; y < 4; ++y) {
//         for(int x=0; x < 4; ++x) {
//             if(offset_y + y + 64 < M) {
//                 if (offset_x + x + 64 < N) {
//                     c_ptr[y * ldc + x]             = beta * c_ptr[y * ldc + x]             + c_penal[y][x];
//                     c_ptr[y * ldc + x + 64]        = beta * c_ptr[y * ldc + x + 64]        + c_penal[y][x+4];
//                     c_ptr[(y + 64) * ldc + x]      = beta * c_ptr[(y + 64) * ldc + x]      + c_penal[y+4][x];
//                     c_ptr[(y + 64) * ldc + x + 64] = beta * c_ptr[(y + 64) * ldc + x + 64] + c_penal[y+4][x + 4];
//                 } else {
//                     if (offset_x + x < N) {
//                         c_ptr[y * ldc + x]        = beta * c_ptr[y * ldc + x]        + c_penal[y][x];
//                         c_ptr[(y + 64) * ldc + x] = beta * c_ptr[(y + 64) * ldc + x] + c_penal[y+4][x];
//                     }
//                 }
//             } else {
//                 if (offset_y + y < M) {
//                     if (offset_x + x + 64 < N) {
//                         c_ptr[y * ldc + x]      = beta * c_ptr[y * ldc + x]      + c_penal[y][x];
//                         c_ptr[y * ldc + x + 64] = beta * c_ptr[y * ldc + x + 64] + c_penal[y][x+4];
//                     } else {
//                         if (offset_x + x < N) {
//                             c_ptr[y * ldc + x] = beta * c_ptr[y * ldc + x] + c_penal[y][x];
//                         }
//                     }
//                 }
//             }
//         }
//     }
// } 

#define SMEM_LDA (132)
#define SMEM_LDB (128)
#define SMEM_LDC (64)

// remove original guard
__device__ __forceinline__ void ldg32_nc_0(float &reg, const void *ptr) {
    asm volatile("{.reg .pred p;\n"
               "mov.b32 %0, 0;\n"
#if __CUDACC_VER_MAJOR__ >= 11 && __CUDACC_VER_MINOR__ >= 4 &&                 \
    __CUDA_ARCH__ >= 750
               "ld.global.nc.L2::128B.f32 %0, [%1];}\n"
#else
               "ld.global.nc.f32 %0, [%1];}\n"
#endif
               : "=f"(reg)
               : "l"(ptr));
}

__device__ __forceinline__ uint32_t smem_u32addr(const void *smem_ptr) {
  uint32_t addr;
  asm("{.reg .u64 u64addr;\n"
      " cvta.to.shared.u64 u64addr, %1;\n"
      " cvt.u32.u64 %0, u64addr;}\n"
      : "=r"(addr)
      : "l"(smem_ptr));

  return addr;
}

__device__ __forceinline__ void lds128(float &reg0, float &reg1, float &reg2,
                                       float &reg3, const uint32_t &addr) {
  asm volatile("ld.shared.v4.f32 {%0, %1, %2, %3}, [%4];\n"
               : "=f"(reg0), "=f"(reg1), "=f"(reg2), "=f"(reg3)
               : "r"(addr));
}

__device__ __forceinline__ void sts128(const float &reg0, const float &reg1,
                                       const float &reg2, const float &reg3,
                                       const uint32_t &addr) {
  asm volatile("st.shared.v4.f32 [%0], {%1, %2, %3, %4};\n"
               :
               : "r"(addr), "f"(reg0), "f"(reg1), "f"(reg2), "f"(reg3));
}

__device__ __forceinline__ void stg128(const float &reg0, const float &reg1,
                                       const float &reg2, const float &reg3,
                                       const float *addr) {
  asm volatile("st.global.v4.f32 [%0], {%1, %2, %3, %4};\n"
               :
               : "l"(addr), "f"(reg0), "f"(reg1), "f"(reg2), "f"(reg3));
}

__device__ __forceinline__ void stg32(const float &reg, const void *ptr) {
  asm volatile("{.reg .pred p;\n"
               " st.global.f32 [%0], %1;}\n"
               :
               : "l"(ptr), "f"(reg));
}

__device__ __forceinline__ void sts32(const float &reg, const uint32_t &addr) {
  asm volatile("st.shared.f32 [%0], %1;\n" : : "r"(addr), "f"(reg));
}

// MY_MMult = [
// 1024 16561.20 7.247925e-05 
// 2048 18817.16 1.525879e-04 
// 3072 18516.94 2.288818e-04 
// 4096 18292.37 4.425049e-04 
// ];
/**
 * version 12 相对于  version  11, 增加 subk 计算中的 ping-pong
 */
__global__ __launch_bounds__(256, 2) void sgemm_128x128x8(int m, int n, int k,
                                                          const float *a,
                                                          const float *b,
                                                          float *c) {

  __shared__ __align__(
      16 * 1024) char smem[24 * 1024]; // 16KB shared memory for buffer

  float *ashare = reinterpret_cast<float *>(smem);
  float *bshare =
      reinterpret_cast<float *>(smem + 16 * 1024); // 8k shared mem for B
  float sum[8][8] = {0};
  float panelA[2][8] = {0}, panelB[2][8] = {0};

  int from_a = (blockIdx.y * 128 + threadIdx.x / 8 * 4) * k + threadIdx.x % 8;
  int from_b = (threadIdx.x / 32) * n + blockIdx.x * 128 + threadIdx.x % 32;

  float a_ldg_reg[4], b_ldg_reg[4];

  uint32_t a_sts_addr = smem_u32addr(ashare + (threadIdx.x % 8) * SMEM_LDA +
                                     (threadIdx.x / 8) * 4);
  uint32_t b_sts_addr =
      smem_u32addr(bshare + (threadIdx.x / 32) * SMEM_LDB + (threadIdx.x % 32));

  uint32_t aptr_base = smem_u32addr(ashare + (threadIdx.x / 16) * 4);
  uint32_t bptr_base = smem_u32addr(bshare + (threadIdx.x % 16) * 4);

  {
// load first
// load gmem to smem for ashare
#pragma unroll
    for (int i = 0; i < 4; ++i) {
      ldg32_nc_0(a_ldg_reg[i],
                 (const char *)(a + from_a) + i * k * sizeof(float));
    }
    sts128(a_ldg_reg[0], a_ldg_reg[1], a_ldg_reg[2], a_ldg_reg[3], a_sts_addr);

// load gmem to smem for bshare
#pragma unroll
    for (int i = 0; i < 4; ++i) {
      ldg32_nc_0(b_ldg_reg[i],
                 (const char *)(b + from_b) + i * 32 * sizeof(float));
    }
#pragma unroll
    for (int i = 0; i < 4; ++i) {
      sts32(b_ldg_reg[i], b_sts_addr + i * 32 * sizeof(float));
    }
    __syncthreads();
    // add offset and flip flag
    from_a += 8;
    from_b += 8 * n;

    a_sts_addr ^= 0x2000;
    b_sts_addr ^= 0x1000;
  }

  // load fisrt panel
  lds128(panelA[0][0], panelA[0][1], panelA[0][2], panelA[0][3], aptr_base);
  lds128(panelA[0][4], panelA[0][5], panelA[0][6], panelA[0][7],
         aptr_base + 64 * sizeof(float));

  lds128(panelB[0][0], panelB[0][1], panelB[0][2], panelB[0][3], bptr_base);
  lds128(panelB[0][4], panelB[0][5], panelB[0][6], panelB[0][7],
         bptr_base + 64 * sizeof(float));

  for (int loop = 0; loop < k; loop += 8) {
// calc
#pragma unroll
    for (int subk = 0; subk < 8; ++subk) {

      if (7 == subk && loop < k - 8) {
        // if have more, load next
        sts128(a_ldg_reg[0], a_ldg_reg[1], a_ldg_reg[2], a_ldg_reg[3],
               a_sts_addr);

#pragma unroll
        for (int i = 0; i < 4; ++i) {
          sts32(b_ldg_reg[i], b_sts_addr + i * 32 * sizeof(float));
        }
        __syncthreads();
        from_a += 8;
        from_b += 8 * n;

        aptr_base ^= 0x2000;
        bptr_base ^= 0x1000;
        a_sts_addr ^= 0x2000;
        b_sts_addr ^= 0x1000;
      }

      const int pp = (subk + 1) % 2; // ping-pong index
      lds128(panelA[pp][0], panelA[pp][1], panelA[pp][2], panelA[pp][3],
             aptr_base + ((subk + 1) % 8) * SMEM_LDA * sizeof(float));
      lds128(panelA[pp][4], panelA[pp][5], panelA[pp][6], panelA[pp][7],
             aptr_base + (((subk + 1) % 8) * SMEM_LDA + 64) * sizeof(float));

      lds128(panelB[pp][0], panelB[pp][1], panelB[pp][2], panelB[pp][3],
             bptr_base + ((subk + 1) % 8) * SMEM_LDB * sizeof(float));
      lds128(panelB[pp][4], panelB[pp][5], panelB[pp][6], panelB[pp][7],
             bptr_base + (((subk + 1) % 8) * SMEM_LDB + 64) * sizeof(float));

      if (0 == subk && loop < k - 8) {
#pragma unroll
        for (int i = 0; i < 4; ++i) {
          ldg32_nc_0(a_ldg_reg[i],
                     (const char *)(a + from_a) + i * k * sizeof(float));
        }
        // load gmem to smem for bshare
#pragma unroll
        for (int i = 0; i < 4; ++i) {
          ldg32_nc_0(b_ldg_reg[i],
                     (const char *)(b + from_b) + i * 32 * sizeof(float));
        }
      }

#pragma unroll
      for (int i = 0; i < 8; ++i) {
#pragma unroll
        for (int j = 0; j < 8; ++j) {
          sum[i][j] += panelA[subk % 2][i] * panelB[subk % 2][j];
        }
      }
    }
  }

  int write_offset = (blockIdx.y * 128 + (threadIdx.x / 16) * 4) * n +
                     blockIdx.x * 128 + (threadIdx.x % 16) * 4;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    stg128(sum[i][0], sum[i][1], sum[i][2], sum[i][3],
           c + write_offset + i * n);
    stg128(sum[i][4], sum[i][5], sum[i][6], sum[i][7],
           c + write_offset + i * n + 64);
    stg128(sum[i + 4][0], sum[i + 4][1], sum[i + 4][2], sum[i + 4][3],
           c + write_offset + (i + 64) * n);
    stg128(sum[i + 4][4], sum[i + 4][5], sum[i + 4][6], sum[i + 4][7],
           c + write_offset + (i + 64) * n + 64);
  }
}

#undef SMEM_LDA
#undef SMEM_LDB


__global__ void sgemm_nt(int M, int N, int K, 
                        float alpha,
                        const float* A, int lda,
                        const float* B, int ldb,
                        float beta,
                        float* C, int ldc) {
    
}

__global__ void sgemm_tn(int M, int N, int K, 
                         float alpha,
                         const float* A, int lda,
                         const float* B, int ldb,
                         float beta,
                         float* C, int ldc) {

}

__global__ void sgemm_tt(int M, int N, int K, 
                         float alpha,
                         const float* A, int lda,
                         const float* B, int ldb,
                         float beta,
                         float* C, int ldc) {
    
}


} // namespace kernel;


void sgemm_nn_cuda_wrap(int M, int N, int K, 
                        float alpha, 
                        const float* A, int lda, 
                        const float* B, int ldb,
                        float beta,
                        float* C, int ldc) {
    
    constexpr int MB_MN = 128;

    int grid_size_m = (M + MB_MN - 1) / MB_MN;
    int grid_size_n = (N + MB_MN - 1) / MB_MN;
    dim3 gridDim(grid_size_m, grid_size_n);
    
    kernel::sgemm_128x128x8<<<gridDim, 256>>> (M, N, K, A, B, C);

}

} // namespace cuda
} // namespace fastnum